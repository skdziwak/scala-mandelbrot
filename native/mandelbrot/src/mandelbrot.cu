#include "hip/hip_runtime.h"
#include "mandelbrot.hpp"
#include <stdlib.h>
#include <stdio.h>

class Complex {
public:
double r, i;
    __device__ Complex(double r, double i) : r(r), i(i) {

    }
    __device__ Complex operator*(const Complex& other) {
        return Complex(this->r * other.r - this->i * other.i, this->r * other.i + this->i * other.r);
    }
    __device__ Complex operator+(const Complex& other) {
        return Complex(this->r + other.r, this->i + other.i);
    }
    __device__ void operator=(const Complex& other) {
        this->r = other.r;
        this->i = other.i;
    }
    __device__ double dist_squared() {
        return this->i * this->i + this->r * this->r;
    }
};

typedef struct {
    unsigned char r, g, b;
} Pixel;

extern "C" {
    __device__ int getX() {
        return blockIdx.x * blockDim.x + threadIdx.x;
    }

    __device__ int getY() {
        return blockIdx.y * blockDim.y + threadIdx.y;
    }

    __device__ int getIndex2D(int x, int y) {
        return x + y * gridDim.x * blockDim.x;
    }

    __device__ int getWidth() {
        return gridDim.x * blockDim.x;
    }

    __device__ int getHeight() {
        return gridDim.y * blockDim.y;
    }

    __device__ Pixel get_color(char* buffer, int x, int y) {
        Pixel p;
        memcpy(&p, &buffer[getIndex2D(x, y) * 3], sizeof(Pixel));
        return p;
    }

    __device__ void set_color(char* buffer, int x, int y, Pixel p) {
        memcpy(&buffer[getIndex2D(x, y) * 3], &p, sizeof(Pixel));
    }

    __device__ int stability(double xc, double yc, int iterations) {
        Complex c(xc, yc);
        Complex p(0.0, 0.0);
        for (int i = 0; i < iterations; i++) {
            p = p * p + c;
            if (p.dist_squared() > 4.0) return i;
        }
        return -1;
    }

    __global__ void mandelbrot(char *dest, FrameParams *params, unsigned char *colors) {
        const int x = getX();
        const int y = getY();
        const int i = getIndex2D(x, y);

        const double width = gridDim.x * blockDim.x;
        const double height = gridDim.y * blockDim.y;
        const double max_dim = width > height ? width : height;
        const double cx = (x - width / 2) / (0.5 * params->zoom * max_dim) + params->x;
        const double cy = (y - height / 2) / (0.5 * params->zoom * max_dim) + params->y;

        int s = stability(cx, cy, params->iterations);

        int index;
        if (s == -1) {
            index = 0;
        } else {
            index = s % 255;
        }

        dest[i * 3] = colors[index * 3];
        dest[i * 3 + 1] = colors[index * 3 + 1];
        dest[i * 3 + 2] = colors[index * 3 + 2];
    }

    __global__ void blur(char* dest, char *src, int r) {
        const static double filter[3][3] = {
            {1.0 / 96, 2.0 / 96, 1.0 / 96},
            {2.0 / 96, 84.0 / 96, 2.0 / 96},
            {1.0 / 96, 2.0 / 96, 1.0 / 96}
        };

        const int x = getX();
        const int y = getY();
        const int i = getIndex2D(x, y);

        const double width = gridDim.x * blockDim.x;
        const double height = gridDim.y * blockDim.y;

        Pixel p;
        p.r = 0;
        p.g = 0;
        p.b = 0;

        for (int dx = -r; dx <= r; dx++) {
            for (int dy = -r; dy <= r; dy++) {
                if (x + dx >= 0 && x + dx < width && y + dy >= 0 && y + dy < height) {
                    Pixel p2 = get_color(src, x + dx, y + dy);
                    p.r += p2.r * filter[dx + r][dy + r];
                    p.g += p2.g * filter[dx + r][dy + r];
                    p.b += p2.b * filter[dx + r][dy + r];
                }
            }
        }

        set_color(dest, x, y, p);
    }

    void render_frame(FrameParams *frame, Render *render, GraphicMemory *graphic_memory) {
        dim3 block(render->block_width, render->block_height);
        dim3 grid(render->grid_width, render->grid_height);
        hipMemcpy(graphic_memory->frame, frame, sizeof(FrameParams), hipMemcpyHostToDevice);
        mandelbrot<<<grid, block>>>(graphic_memory->buffer, graphic_memory->frame, graphic_memory->colors);
        blur<<<grid, block>>>(graphic_memory->buffer2, graphic_memory->buffer, 1);
        hipMemcpy(render->buffer, graphic_memory->buffer2, graphic_memory->size, hipMemcpyDeviceToHost);
    }

    GraphicMemory *create_graphic_memory(Render *render) {
        printf("Creating graphic memory\n");
        GraphicMemory *graphic_memory = (GraphicMemory*) malloc(sizeof(GraphicMemory));
        size_t size = sizeof(char) * render->grid_height * render->grid_width * render->block_width * render->block_height * 3;
        graphic_memory->size = size;
        if (hipMalloc(&graphic_memory->buffer, size) != hipSuccess) {
            printf("Error: %s", hipGetErrorString(hipGetLastError()));
            return nullptr;
        }
        if (hipMalloc(&graphic_memory->buffer2, size) != hipSuccess) {
            printf("Error: %s", hipGetErrorString(hipGetLastError()));
            return nullptr;
        }
        if (hipMalloc(&graphic_memory->colors, sizeof(char) * 256 * 3) != hipSuccess) {
            printf("Error: %s", hipGetErrorString(hipGetLastError()));
            return nullptr;
        }
        if (hipMalloc(&graphic_memory->frame, sizeof(FrameParams)) != hipSuccess) {
            printf("Error: %s", hipGetErrorString(hipGetLastError()));
            return nullptr;
        }
        printf("Allocated graphic memory\n");
        hipMemcpy(graphic_memory->colors, render->colors, sizeof(char) * 256 * 3, hipMemcpyHostToDevice);
        return graphic_memory;
    }

    void free_graphic_memory(GraphicMemory *graphic_memory) {
        printf("Freeing graphic memory\n");
        hipFree(graphic_memory->buffer);
        hipFree(graphic_memory->buffer2);
        hipFree(graphic_memory->colors);
        hipFree(graphic_memory->frame);
        free(graphic_memory);
    }

    Render* create_render(int block_width, int block_height, int grid_width, int grid_height) {
        printf("Creating render\n");
        Render *render = (Render*) malloc(sizeof(Render));
        render->block_width = block_width;
        render->block_height = block_height;
        render->grid_width = grid_width;
        render->grid_height = grid_height;
        render->buffer = (char*) malloc(sizeof(char) * block_width * block_height * grid_width * grid_height * 3);
        render->colors = (unsigned char*) malloc(sizeof(char) * 256 * 3);
        if (render->buffer == nullptr || render->colors == nullptr) {
            return nullptr;
        }
        return render;
    }

    void free_render(Render *render) {
        printf("Freeing render\n");
        free(render->buffer);
        free(render->colors);
        free(render);
    }


    char* copy_preview(Render *render) {
        size_t size = sizeof(char) * render->block_width * render->block_height * render->grid_width * render->grid_height * 3;
        char *preview = (char*) malloc(size);
        memcpy(preview, render->buffer, size);
        return preview;
    }

    void free_preview(char *preview) {
        free(preview);
    }
}

extern "C" int assert_42_value() {
    return 42;
}